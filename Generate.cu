#include "hip/hip_runtime.h"
#include "framework.h"
#include "AccumData.h"

__device__ void transform(ITERATOR* iter);

__device__ __forceinline__ float atomicMinFloat(float* addr, float value) {
	float old;
	old = (value >= 0) ? __int_as_float(atomicMin((int*)addr, __float_as_int(value))) :
		__uint_as_float(atomicMax((unsigned int*)addr, __float_as_uint(value)));

	return old;
}

__device__ __forceinline__ float atomicMaxFloat(float* addr, float value) {
	float old;
	old = (value >= 0) ? __int_as_float(atomicMax((int*)addr, __float_as_int(value))) :
		__uint_as_float(atomicMin((unsigned int*)addr, __float_as_uint(value)));

	return old;
}

__global__ void iterate(ACCUM_PARAMS params, GPU_ARRAY_2D arrIter, GPU_ARRAY_2D arrAccum, PVOID pStats)
{
	UINT idx = blockIdx.x * blockDim.x + threadIdx.x;
	ITERATOR* iter = &(((ITERATOR*)(arrIter.pArray))[idx]);
	ACCUM_STATS* accumStats = (ACCUM_STATS*)pStats;

	if (accumStats->bAbort) return;
	
	// On intialize, seed the random number generators
	if (params.bInit)
	{
		iter->rand.init(idx);
	}

	for (UINT i = 0; i < params.nSteps; i++)
	{
		transform(iter);

		if (!isfinite(iter->x) || !isfinite(iter->y))
		{
			accumStats->bAbort = TRUE;
			break;
		}

		if (!params.bInit)
		{
			int i = int(iter->x * params.rect.fScale + params.rect.fOffsetX);
			int j = int(iter->y * params.rect.fScale + params.rect.fOffsetY);
			if (i >= 0 && i < (int)arrAccum.nWidth && j >= 0 && j < (int)arrAccum.nHeight)
			{
				FLOAT_COLOR* element = (FLOAT_COLOR*)((unsigned char *)(arrAccum.pArray) + j * arrAccum.nPitch + i * sizeof(FLOAT_COLOR));
				if (params.bHitPercent)
				{
					atomicAdd(&(accumStats->nHitRect), 1);
					if (element->IsZero())
						atomicAdd(&(accumStats->nNewHits), 1);
				}
				atomicAdd(&(element->r), iter->clr.r);
				atomicAdd(&(element->g), iter->clr.g);
				atomicAdd(&(element->b), iter->clr.b);
				atomicMax(&(accumStats->nMaxColorElement), (UINT)element->Max());
			}
		}
	}

	// On initialize, adjust the bounding box based on the zeroth block
	if (params.bInit && (blockIdx.x == 0))
	{
		atomicMaxFloat(&(accumStats->xMax), iter->x);
		atomicMaxFloat(&(accumStats->yMax), iter->y);
		atomicMinFloat(&(accumStats->xMin), iter->x);
		atomicMinFloat(&(accumStats->yMin), iter->y);
	}
}

__device__ void transform(ITERATOR* iter)
{
	float rnd = iter->rand.frand();
	FLOAT_COLOR clr;
	if (rnd < 0.33333f)
	{
		iter->x = iter->x * 0.5f;
		iter->y = iter->y * 0.5f + 0.5f;
		clr.r = 1.0f;
		clr.g = 1.0f;
		clr.b = 0.0f;
	}
	else if (rnd < 0.66666f)
	{
		iter->x = iter->x * 0.5f + 0.433f;
		iter->y = iter->y * 0.5f - 0.25f;
		clr.r = 1.0f;
		clr.g = 0.0f;
		clr.b = 1.0f;
	}
	else
	{
		iter->x = iter->x * 0.5f - 0.433f;
		iter->y = iter->y * 0.5f - 0.25f;
		clr.r = 0.0f;
		clr.g = 1.0f;
		clr.b = 1.0f;
	}
	iter->clr.Tint(clr, 3.0f);
}

__global__ void render_texture(const RENDER_PARAMS params, GPU_ARRAY_2D texture, GPU_ARRAY_2D arrAccum)
{
	UINT texx = blockIdx.x * blockDim.x + threadIdx.x;
	UINT texy = blockIdx.y * blockDim.y + threadIdx.y;
	if (texx >= texture.nWidth || texy >= texture.nHeight) return;

	float *pixel = (float*)((unsigned char *)(texture.pArray) + texy * texture.nPitch) + 4 * texx;

	int iAntiAlias = max(1, params.iAntiAlias);
	UINT arrx = texx * iAntiAlias;
	UINT arry = texy * iAntiAlias;

	float r = 0.0f, g = 0.0f, b = 0.0f;
	for (UINT j = 0; j < iAntiAlias; j++)
	{
		FLOAT_COLOR* pRow = (FLOAT_COLOR*)((unsigned char *)arrAccum.pArray + (arry + j) * arrAccum.nPitch);
		for (UINT i = 0; i < iAntiAlias; i++)
		{
			FLOAT_COLOR* pItem = &pRow[arrx + i];
			if(pItem->r) r += logf(pItem->r) * params.fLogColorScale;
			if(pItem->b) b += logf(pItem->b) * params.fLogColorScale;
			if(pItem->g) g += logf(pItem->g) * params.fLogColorScale;
		}
	}
	pixel[0] = r / (float)(iAntiAlias * iAntiAlias);
	pixel[1] = g / (float)(iAntiAlias * iAntiAlias);
	pixel[2] = b / (float)(iAntiAlias * iAntiAlias);
	pixel[3] = 1.0f;
}

hipError_t cuda_iterate(const ACCUM_PARAMS& params, GPU_ARRAY_2D& arrIter, GPU_ARRAY_2D& arrAccum, PVOID pStats)
{
	hipError_t error = hipSuccess;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipEventRecord(start);
	iterate << < arrIter.nWidth, arrIter.nHeight >> > (params, arrIter, arrAccum, pStats);
	hipEventRecord(stop);
	error = hipGetLastError();
	if (error != hipSuccess) return error;

	hipEventSynchronize(stop);
	error = hipGetLastError();
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);

	return error;
}

hipError_t cuda_render_texture(const RENDER_PARAMS& params, GPU_ARRAY_2D& texture, GPU_ARRAY_2D& arrAccum)
{
	hipError_t error = hipSuccess;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	dim3 Db = dim3(16, 16);   
	dim3 Dg = dim3(((UINT)texture.nWidth + Db.x - 1) / Db.x, ((UINT)texture.nHeight + Db.y - 1) / Db.y);

	hipEventRecord(start);
	render_texture <<<Dg, Db>>> (params, texture, arrAccum);
	hipEventRecord(stop);
	error = hipGetLastError();
	if (error != hipSuccess) return error;

	hipEventSynchronize(stop);
	error = hipGetLastError();
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);

	return error;
}

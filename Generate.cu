#include "hip/hip_runtime.h"
#include "framework.h"
#include "AccumData.h"
#include "Transform.h"
#include "CudaArray.h"

__device__ void transform(const CCudaArray1D<CTransform> &arrTransforms, ITERATOR* iter);

__device__ __forceinline__ float atomicMinFloat(float* addr, float value) {
	float old;
	old = (value >= 0) ? __int_as_float(atomicMin((int*)addr, __float_as_int(value))) :
		__uint_as_float(atomicMax((unsigned int*)addr, __float_as_uint(value)));

	return old;
}

__device__ __forceinline__ float atomicMaxFloat(float* addr, float value) {
	float old;
	old = (value >= 0) ? __int_as_float(atomicMax((int*)addr, __float_as_int(value))) :
		__uint_as_float(atomicMin((unsigned int*)addr, __float_as_uint(value)));

	return old;
}

__global__ void iterate(ACCUM_PARAMS params, CCudaArray1D<CTransform> arrTransforms, CCudaArray1D<ITERATOR> arrIter, CCudaArray2D<ACCUM> arrAccum, PVOID pStats)
{
	UINT idx = blockIdx.x * blockDim.x + threadIdx.x;
	ITERATOR* iter = arrIter.GetAt(idx);
	ACCUM_STATS* accumStats = (ACCUM_STATS*)pStats;

	if (accumStats->bAbort) return;
	
	// On intialize, seed the random number generators
	if (params.bInit)
	{
		iter->rand.init(idx);
	}

	for (UINT i = 0; i < params.nSteps; i++)
	{
		transform(arrTransforms, iter);

		if (!isfinite(iter->pos[0]) || !isfinite(iter->pos[1]))
		{
			accumStats->bAbort = TRUE;
			break;
		}

		if (!params.bInit)
		{
			int i = int(iter->pos[0] * params.rect.fScale + params.rect.fOffsetX);
			int j = int(iter->pos[1] * params.rect.fScale + params.rect.fOffsetY);
			if (arrAccum.ValidIndex(i, j))
			{
				ACCUM* element = arrAccum.GetAt(i, j);
				UINT nCount = atomicAdd(&(element->nCount), 1);
				atomicMax(&(accumStats->nMaxCount), nCount);
				if (params.bHitPercent)
				{
					atomicAdd(&(accumStats->nHitRect), 1);
					if (nCount == 0)
						atomicAdd(&(accumStats->nNewHits), 1);
				}
				atomicAdd(&(element->clrAccum.r), iter->clr.r);
				atomicAdd(&(element->clrAccum.g), iter->clr.g);
				atomicAdd(&(element->clrAccum.b), iter->clr.b);
				atomicMaxFloat(&(accumStats->fMaxColorElement), element->clrAccum.Max());
			}
		}
	}

	// On initialize, adjust the bounding box based on the zeroth block
	if (params.bInit && (blockIdx.x == 0))
	{
		atomicMaxFloat(&(accumStats->xMax), iter->pos[0]);
		atomicMaxFloat(&(accumStats->yMax), iter->pos[1]);
		atomicMinFloat(&(accumStats->xMin), iter->pos[0]);
		atomicMinFloat(&(accumStats->yMin), iter->pos[1]);
	}
}

__device__ void transform(const CCudaArray1D<CTransform> &arrTransforms, ITERATOR* iter)
{
	float rnd = iter->rand.frand();
	for (UINT i = 0; i < arrTransforms.Length(); i++)
	{
		const CTransform* pTrans = arrTransforms.GetAt(i);
		if (rnd <= pTrans->Weight())
		{
			iter->pos = pTrans->Matrix0() * iter->pos;
			iter->pos += pTrans->Offset0();
			iter->clr.Tint(pTrans->Color(), 3.0f);
			break;
		}
	}
}

hipError_t cuda_iterate(const ACCUM_PARAMS& params, CCudaArray1D<CTransform>& arrTransforms, CCudaArray1D<ITERATOR>& arrIter, CCudaArray2D<ACCUM>& arrAccum, PVOID pStats)
{
	assert(params.nBlocks * params.nThreads == arrIter.Length());
	hipError_t error = hipSuccess;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipEventRecord(start);
	iterate << < params.nBlocks, params.nThreads >> > (params, arrTransforms, arrIter, arrAccum, pStats);
	hipEventRecord(stop);
	error = hipGetLastError();
	if (error != hipSuccess) return error;

	hipEventSynchronize(stop);
	error = hipGetLastError();
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);

	return error;
}


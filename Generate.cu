#include "hip/hip_runtime.h"
#include "framework.h"
#include "AccumData.h"
#include "randgen.h"

__global__ void initialize(ACCUM_PARAMS params, GPU_ARRAY_2D randgen, PVOID pStats)
{
	UINT idx = blockIdx.x * blockDim.x + threadIdx.x;
	CRandgen* rand = &(((CRandgen*)(randgen.pArray))[idx]);

	// On intialize, seed the random number generators
	if (params.bInit)
	{
		rand->init(idx);
	}
}

__global__ void test_iterate(ACCUM_PARAMS params, GPU_ARRAY_2D randgen, GPU_ARRAY_2D arrAccum, PVOID pStats)
{
	UINT tidx = blockIdx.x * blockDim.x + threadIdx.x;
	UINT tidy = blockIdx.y * blockDim.y + threadIdx.y;
	CRandgen *rand = &(((CRandgen*)(randgen.pArray))[threadIdx.x + threadIdx.y * randgen.nWidth]);
	if (tidx >= arrAccum.nWidth || tidy >= arrAccum.nHeight) return;
	unsigned char* pArray = (unsigned char *)(arrAccum.pArray);
	
	COUNT_COLOR *element = (COUNT_COLOR*)(pArray + tidy * arrAccum.nPitch + tidx * sizeof(COUNT_COLOR));
	element->nCount = tidx + tidy + 2;
	element->r = rand->frand();
	element->g = rand->frand();
	element->b = rand->frand();
	atomicMax(&((ACCUM_STATS *)pStats)->nMaxCount, element->nCount);
}

__global__ void render_texture(const RENDER_PARAMS params, GPU_ARRAY_2D texture, GPU_ARRAY_2D arrAccum)
{
	UINT texx = blockIdx.x * blockDim.x + threadIdx.x;
	UINT texy = blockIdx.y * blockDim.y + threadIdx.y;
	if (texx >= texture.nWidth || texy >= texture.nHeight) return;

	float *pixel = (float*)((unsigned char *)(texture.pArray) + texy * texture.nPitch) + 4 * texx;

	int iAntiAlias = max(1, params.iAntiAlias);
	UINT arrx = texx * iAntiAlias;
	UINT arry = texy * iAntiAlias;

	float r = 0.0f, g = 0.0f, b = 0.0f, a = 0.0f;
	for (UINT j = 0; j < iAntiAlias; j++)
	{
		COUNT_COLOR* pRow = (COUNT_COLOR*)((unsigned char *)arrAccum.pArray + (arry + j) * arrAccum.nPitch);
		for (UINT i = 0; i < iAntiAlias; i++)
		{
			COUNT_COLOR* pItem = &pRow[arrx + i];
			a += (float)(pItem->nCount) * params.fCountScale;
			r += pItem->r;
			g += pItem->g;
			b += pItem->b;
		}
	}
	a /= (float)(iAntiAlias * iAntiAlias);
	pixel[0] = r / (float)(iAntiAlias * iAntiAlias) * a;
	pixel[1] = g / (float)(iAntiAlias * iAntiAlias) * a;
	pixel[2] = b / (float)(iAntiAlias * iAntiAlias) * a;
	pixel[3] = 1.0f;
}

hipError_t cuda_intialize(const ACCUM_PARAMS& params, GPU_ARRAY_2D& randgen, PVOID pStats)
{
	hipError_t error = hipSuccess;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipEventRecord(start);
	initialize << < randgen.nWidth, randgen.nHeight >> > (params, randgen, pStats);
	hipEventRecord(stop);
	error = hipGetLastError();
	if (error != hipSuccess) return error;

	hipEventSynchronize(stop);
	error = hipGetLastError();
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);

	return error;
}

hipError_t cuda_test_generate(const ACCUM_PARAMS& params, GPU_ARRAY_2D& randgen, GPU_ARRAY_2D& arrAccum, PVOID pStats)
{
	hipError_t error = hipSuccess;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	dim3 Db = dim3(randgen.nWidth, randgen.nHeight);   
	dim3 Dg = dim3(((UINT)arrAccum.nWidth + Db.x - 1) / Db.x, ((UINT)arrAccum.nHeight + Db.y - 1) / Db.y);

	hipEventRecord(start);
	test_iterate << <Dg, Db >> > (params, randgen, arrAccum, pStats);
	hipEventRecord(stop);
	error = hipGetLastError();
	if (error != hipSuccess) return error;

	hipEventSynchronize(stop);
	error = hipGetLastError();
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);

	return error;
}

hipError_t cuda_render_texture(const RENDER_PARAMS& params, GPU_ARRAY_2D& texture, GPU_ARRAY_2D& arrAccum)
{
	hipError_t error = hipSuccess;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	dim3 Db = dim3(16, 16);   
	dim3 Dg = dim3(((UINT)texture.nWidth + Db.x - 1) / Db.x, ((UINT)texture.nHeight + Db.y - 1) / Db.y);

	hipEventRecord(start);
	render_texture <<<Dg, Db>>> (params, texture, arrAccum);
	hipEventRecord(stop);
	error = hipGetLastError();
	if (error != hipSuccess) return error;

	hipEventSynchronize(stop);
	error = hipGetLastError();
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);

	return error;
}

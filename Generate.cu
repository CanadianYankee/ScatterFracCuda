#include "hip/hip_runtime.h"
#include "framework.h"
#include "AccumData.h"

__device__ void transform(ITERATOR* iter);

__device__ __forceinline__ float atomicMinFloat(float* addr, float value) {
	float old;
	old = (value >= 0) ? __int_as_float(atomicMin((int*)addr, __float_as_int(value))) :
		__uint_as_float(atomicMax((unsigned int*)addr, __float_as_uint(value)));

	return old;
}

__device__ __forceinline__ float atomicMaxFloat(float* addr, float value) {
	float old;
	old = (value >= 0) ? __int_as_float(atomicMax((int*)addr, __float_as_int(value))) :
		__uint_as_float(atomicMin((unsigned int*)addr, __float_as_uint(value)));

	return old;
}

__global__ void iterate(ACCUM_PARAMS params, GPU_ARRAY_2D arrIter, GPU_ARRAY_2D arrAccum, PVOID pStats)
{
	UINT idx = blockIdx.x * blockDim.x + threadIdx.x;
	ITERATOR* iter = &(((ITERATOR*)(arrIter.pArray))[idx]);
	ACCUM_STATS* accumStats = (ACCUM_STATS*)pStats;

	if (accumStats->bAbort) return;
	
	// On intialize, seed the random number generators
	if (params.bInit)
	{
		iter->rand.init(idx);
	}

	for (UINT i = 0; i < params.nSteps; i++)
	{
		transform(iter);

		if (!isfinite(iter->x) || !isfinite(iter->y))
		{
			accumStats->bAbort = TRUE;
			break;
		}

		if (!params.bInit)
		{
			int i = int(iter->x * params.rect.fScale + params.rect.fOffsetX);
			int j = int(iter->y * params.rect.fScale + params.rect.fOffsetY);
			if (i >= 0 && i < (int)arrAccum.nWidth && j >= 0 && j < (int)arrAccum.nHeight)
			{
				ACCUM* element = (ACCUM*)((unsigned char *)(arrAccum.pArray) + j * arrAccum.nPitch + i * sizeof(ACCUM));
				UINT nCount = atomicAdd(&(element->nCount), 1);
				atomicMax(&(accumStats->nMaxCount), nCount);
				if (params.bHitPercent)
				{
					atomicAdd(&(accumStats->nHitRect), 1);
					if (nCount == 0)
						atomicAdd(&(accumStats->nNewHits), 1);
				}
				atomicAdd(&(element->clrAccum.r), iter->clr.r);
				atomicAdd(&(element->clrAccum.g), iter->clr.g);
				atomicAdd(&(element->clrAccum.b), iter->clr.b);
				atomicMax(&(accumStats->nMaxColorElement), (UINT)ceil(element->clrAccum.Max()));
			}
		}
	}

	// On initialize, adjust the bounding box based on the zeroth block
	if (params.bInit && (blockIdx.x == 0))
	{
		atomicMaxFloat(&(accumStats->xMax), iter->x);
		atomicMaxFloat(&(accumStats->yMax), iter->y);
		atomicMinFloat(&(accumStats->xMin), iter->x);
		atomicMinFloat(&(accumStats->yMin), iter->y);
	}
}

__device__ void transform(ITERATOR* iter)
{
	float rnd = iter->rand.frand();
	FLOAT_COLOR clr;
	if (rnd < 0.5f)
	{
		iter->x = iter->x * 0.5f;
		iter->y = iter->y * 0.5f + 0.5f;
		clr.r = 1.0f;
		clr.g = 1.0f;
		clr.b = 0.0f;
	}
	else if (rnd < 0.9f)
	{
		iter->x = iter->x * 0.5f + 0.433f;
		iter->y = iter->y * 0.5f - 0.25f;
		clr.r = 1.0f;
		clr.g = 0.0f;
		clr.b = 1.0f;
	}
	else
	{
		iter->x = iter->x * 0.5f - 0.433f;
		iter->y = iter->y * 0.5f - 0.25f;
		clr.r = 0.0f;
		clr.g = 1.0f;
		clr.b = 1.0f;
	}
	iter->clr.Tint(clr, 3.0f);
}

// Checks for out-of-range and does atomic adds
__device__ inline void AddFiltered(GPU_ARRAY_2D& arrFiltered, int x, int y, const FLOAT_COLOR& clr)
{
	if (x >= 0 && x < (int)(arrFiltered.nWidth) && y >= 0 && y < (int)(arrFiltered.nHeight))
	{
		FILTERED* pFiltered = (FILTERED*)((unsigned char*)(arrFiltered.pArray) + y * arrFiltered.nPitch) + x;
		if (clr.r) atomicAdd(&(pFiltered->r), clr.r);
		if (clr.g) atomicAdd(&(pFiltered->g), clr.g);
		if (clr.b) atomicAdd(&(pFiltered->b), clr.b);
	}
}

__device__ inline float gaussian(float x, float stddev)
{
	float x0 = x / stddev;
	return exp(-0.5f * x0 * x0) / sqrt(2.0f * stddev);
}

__global__ void rescale_filter(const RENDER_PARAMS params, GPU_ARRAY_2D arrFiltered, GPU_ARRAY_2D arrAccum)
{
	int arrx = blockIdx.x * blockDim.x + threadIdx.x;
	int arry = blockIdx.y * blockDim.y + threadIdx.y;
	if (arrx >= arrAccum.nWidth || arry >= arrAccum.nHeight) return;

	ACCUM* pAccum = (ACCUM*)((unsigned char*)(arrAccum.pArray) + arry * arrAccum.nPitch + arrx * sizeof(ACCUM));
	int rad = params.iKernelRadius * params.iAntiAlias;
	int fx = arrx - rad;
	int fy = arry - rad;
	float fCutoff = rad ? 0.001f / (float)(rad * rad) : 0.0f;
	if (!pAccum->clrAccum.IsZero())
	{
		FLOAT_COLOR clr = pAccum->clrAccum;
		clr.LogScale(params.fLogColorScale);
		if (!clr.IsZero())
		{
			float h, s, v;
			clr.ToHSV(h, s, v);
			v = powf(v, params.fValuePower);
			if (params.fSaturPower) s = powf(s, params.fSaturPower);
			clr.FromHSV(h, s, v);
			if (rad)
			{
				float fHalfRad = 0.5f * (float)rad;
				float stddev = params.fFilterScale / pow((float)(pAccum->nCount), params.fKernelAlpha);
				if (stddev > fHalfRad) stddev = fHalfRad;
				if (stddev < 0.5f)
				{
					// No dispersion
					AddFiltered(arrFiltered, fx, fy, clr);
				}
				else
				{
					// Calculate one-eighth(ish) of the filter and use symmetry to get the rest
					for (int ix = 0; ix <= rad; ix++)
					{
						float attx = gaussian((float)ix, stddev);
						for (int iy = 0; iy <= ix; iy++)
						{
							float att = attx * gaussian((float)iy, stddev);
							if (att < fCutoff) continue;	// Don't bother if numbers are tiny
							FLOAT_COLOR attClr = att * clr;
							AddFiltered(arrFiltered, fx + ix, fy + iy, attClr);
							if (ix || iy)
							{
								if (iy == 0)
								{
									AddFiltered(arrFiltered, fx - ix, fy, attClr);
									AddFiltered(arrFiltered, fx, fy + ix, attClr);
									AddFiltered(arrFiltered, fx, fy - ix, attClr);
								}
								else
								{
									AddFiltered(arrFiltered, fx - ix, fy + iy, attClr);
									AddFiltered(arrFiltered, fx + ix, fy - iy, attClr);
									AddFiltered(arrFiltered, fx - ix, fy - iy, attClr);
									if (ix != iy)
									{
										AddFiltered(arrFiltered, fx + iy, fy + ix, attClr);
										AddFiltered(arrFiltered, fx - iy, fy + ix, attClr);
										AddFiltered(arrFiltered, fx + iy, fy - ix, attClr);
										AddFiltered(arrFiltered, fx - iy, fy - ix, attClr);
									}
								}
							}
						}
					}
				}
			}
			else
			{
				// Just scaling, no actual filtering
				FILTERED* pFiltered = (FILTERED*)((unsigned char*)(arrFiltered.pArray) + fy * arrFiltered.nPitch + fx * sizeof(FILTERED));
				*pFiltered = clr;
			}
		}
	}
}

__global__ void render_texture(const RENDER_PARAMS params, GPU_ARRAY_2D texture, GPU_ARRAY_2D arrFiltered)
{
	UINT texx = blockIdx.x * blockDim.x + threadIdx.x;
	UINT texy = blockIdx.y * blockDim.y + threadIdx.y;
	if (texx >= texture.nWidth || texy >= texture.nHeight) return;

	float *pixel = (float*)((unsigned char *)(texture.pArray) + texy * texture.nPitch) + 4 * texx;

	UINT iAntiAlias = max(1, params.iAntiAlias);
	UINT arrx = texx * iAntiAlias;
	UINT arry = texy * iAntiAlias;

	float r = 0.0f, g = 0.0f, b = 0.0f;
	for (UINT j = 0; j < iAntiAlias; j++)
	{
		FILTERED* pRow = (FILTERED*)((unsigned char *)arrFiltered.pArray + (arry + j) * arrFiltered.nPitch);
		for (UINT i = 0; i < iAntiAlias; i++)
		{
			FILTERED* pItem = &pRow[arrx + i];
			if (!pItem->IsZero())
			{
				r += pItem->r;
				g += pItem->g;
				b += pItem->b;
			}
		}
	}
	pixel[0] = r / (float)(iAntiAlias * iAntiAlias);
	pixel[1] = g / (float)(iAntiAlias * iAntiAlias);
	pixel[2] = b / (float)(iAntiAlias * iAntiAlias);
	pixel[3] = 1.0f;
}

hipError_t cuda_iterate(const ACCUM_PARAMS& params, GPU_ARRAY_2D& arrIter, GPU_ARRAY_2D& arrAccum, PVOID pStats)
{
	hipError_t error = hipSuccess;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipEventRecord(start);
	iterate << < arrIter.nWidth, arrIter.nHeight >> > (params, arrIter, arrAccum, pStats);
	hipEventRecord(stop);
	error = hipGetLastError();
	if (error != hipSuccess) return error;

	hipEventSynchronize(stop);
	error = hipGetLastError();
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);

	return error;
}

hipError_t cuda_render_texture(const RENDER_PARAMS& params, GPU_ARRAY_2D& texture, GPU_ARRAY_2D& arrFiltered, GPU_ARRAY_2D& arrAccum)
{
	hipError_t error = hipSuccess;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	
	// First run the log-scale and density estimation to go from arrAccum -> arrFiltered
	dim3 Db = dim3(16, 16);   
	dim3 Dg = dim3(((UINT)arrAccum.nWidth + Db.x - 1) / Db.x, ((UINT)arrAccum.nHeight + Db.y - 1) / Db.y);
	error = hipMemset2D(arrFiltered.pArray, arrFiltered.nPitch, 0, arrFiltered.nWidth * sizeof(FILTERED), arrFiltered.nHeight);
	if (error != hipSuccess) return error;

	hipEventRecord(start);
	rescale_filter << <Dg, Db >> > (params, arrFiltered, arrAccum);
	hipEventRecord(stop);
	error = hipGetLastError();
	if (error != hipSuccess) return error;
	hipEventSynchronize(stop);
	error = hipGetLastError();
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
	if (error != hipSuccess) return error;

	// Next, do anti-aliasing and final conversion to texture to go from arrFiltered -> texture
	Dg = dim3(((UINT)texture.nWidth + Db.x - 1) / Db.x, ((UINT)texture.nHeight + Db.y - 1) / Db.y);

	hipEventRecord(start);
	render_texture <<<Dg, Db>>> (params, texture, arrFiltered);
	hipEventRecord(stop);
	error = hipGetLastError();
	if (error != hipSuccess) return error;

	hipEventSynchronize(stop);
	error = hipGetLastError();
	milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);

	return error;
}
